/*
 * Copyright (c) 2016 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual
 * property and proprietary rights in and to this software,
 * related documentation and any modifications thereto.  Any use,
 * reproduction, disclosure or distribution of this software and
 * related documentation without an express license agreement from
 * NVIDIA CORPORATION is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <cstddef>
#include <cinttypes>
#include <hip/device_functions.h>

static inline __device__ float
clamp(const float v, const float low, const float high) {
	return v < low ? low : v > high ? high : v;
}

static inline __device__ float
rgb2y(const uint8_t r, const uint8_t g, const uint8_t b) {
	return 0.299f*(float)r + 0.587f*(float)g + 0.114f*(float)b;
}
static inline __device__ float
rgb2u(const uint8_t r, const uint8_t g, const uint8_t b) {
	const float y = rgb2y(r,g,b);
	return clamp(-(((-(float)b + y) / 1.732446f) - 128.f), 0.f, 255.f);
}
static inline __device__ float
rgb2v(const uint8_t r, const uint8_t g, const uint8_t b) {
	const float y = rgb2y(r,g,b);
	const float u = rgb2u(r,g,b);
	return clamp((y - (0.337633f*(u-128.f)) - (float)g) / 0.698001f + 128.f,
	             0.f, 255.f);
}

/* Converts from RGB data to NV12.  NV12's "U" and "V" channels are interleaved
 * and subsampled 2x2.  Note the RGB data are not pitched. */
extern "C" __global__ void
rgb2yuv(const uint8_t* const __restrict rgb,
        const size_t width, const size_t height, const size_t c/*omponents*/,
        uint8_t* const __restrict yuv, unsigned pitch) {
	const size_t x = blockIdx.x*blockDim.x + threadIdx.x;
	const size_t y = blockIdx.y*blockDim.y + threadIdx.y;
	const size_t i = y*pitch + x;
	const size_t j = y*width + x;
	if(x >= width || y >= height || i >= pitch*height || j >= width*height) {
		return;
	}
	assert(pitch >= width);
	assert(i < pitch*height);
	assert(j < width*height);
	assert(width <= 4096);
	assert(height <= 4096);
	assert(c == 3 || c == 4);
	assert(pitch <= 4096);
	uint8_t* __restrict Y = yuv;
	Y[i] = (uint8_t)clamp(rgb2y(rgb[j*c+0], rgb[j*c+1], rgb[j*c+2]), 0, 255);
	/* U+V are downsampled 2x per dimension.  So kill off 3 of every 4 threads
	 * that reach here; only one will do the writes into U and V. */
	/* thought: use x0 to write into U and x1 to write into V, to spread load? */
	if(x&1 == 1 || y&1 == 1) {
		return;
	}
	uint8_t* __restrict uv = yuv + pitch*height;
	const size_t uvidx = y/2*(pitch/2) + x/2;
	const size_t idx[4] = {
		min((y+0)*width + x+0, width*height),
		min((y+0)*width + x+1, width*height),
		min((y+1)*width + x+0, width*height),
		min((y+1)*width + x+1, width*height),
	};
	const float u[4] = {
		rgb2u(rgb[idx[0]*c+0], rgb[idx[0]*c+1], rgb[idx[0]*c+2]),
		rgb2u(rgb[idx[1]*c+0], rgb[idx[1]*c+1], rgb[idx[1]*c+2]),
		rgb2u(rgb[idx[2]*c+0], rgb[idx[2]*c+1], rgb[idx[2]*c+2]),
		rgb2u(rgb[idx[3]*c+0], rgb[idx[3]*c+1], rgb[idx[3]*c+2])
	};
	const float v[4] = {
		rgb2v(rgb[idx[0]*c+0], rgb[idx[0]*c+1], rgb[idx[0]*c+2]),
		rgb2v(rgb[idx[1]*c+0], rgb[idx[1]*c+1], rgb[idx[1]*c+2]),
		rgb2v(rgb[idx[2]*c+0], rgb[idx[2]*c+1], rgb[idx[2]*c+2]),
		rgb2v(rgb[idx[3]*c+0], rgb[idx[3]*c+1], rgb[idx[3]*c+2])
	};
	uv[uvidx*2+0] = (uint8_t)clamp((u[0] + u[1] + u[2] + u[3]) / 4.0, 0, 255);
	uv[uvidx*2+1] = (uint8_t)clamp((v[0] + v[1] + v[2] + v[3]) / 4.0, 0, 255);
}

static inline __device__ float
yuv2r(const uint8_t y, const uint8_t u, const uint8_t v) {
	(void)u;
	return (y-16)*1.164f + (1.596f * (v-128));
}
static inline __device__ float
yuv2g(const uint8_t y, const uint8_t u, const uint8_t v) {
	return (y-16)*1.164f + (u-128)*-0.392f + (v-128)*-0.813f;
}
static inline __device__ float
yuv2b(const uint8_t y, const uint8_t u, const uint8_t v) {
	(void)v;
	return (y-16)*1.164 + (u-128)*2.017;
}

/* Convert back from NV12 to RGB.   Note the RGB buffer is not pitched. */
extern "C" __global__ void
yuv2rgb(const uint8_t* const __restrict yuv,
        const size_t width, const size_t height, unsigned pitch,
        uint8_t* const __restrict rgb) {
	const size_t x = blockIdx.x*blockDim.x + threadIdx.x;
	const size_t y = blockIdx.y*blockDim.y + threadIdx.y;
	const size_t i = y*pitch + x;
	const size_t j = y*width + x;
	if(x >= width || y >= height || i >= pitch*height || j >= width*height) {
		return;
	}
	assert(i < pitch*height);
	assert(j < width*height);
	assert(width <= 4096);
	assert(height <= 4096);
	assert(pitch <= 4096);
	const uint8_t* __restrict Y = yuv;
	const uint8_t* __restrict uv = yuv + pitch*height;
	const size_t idx[4] = {
		min((y+0)/2*pitch/2 + (x+0)/2, height*pitch),
		min((y+0)/2*pitch/2 + (x+1)/2, height*pitch),
		min((y+1)/2*pitch/2 + (x+0)/2, height*pitch),
		min((y+1)/2*pitch/2 + (x+1)/2, height*pitch),
	};
	const uint8_t u[4] = {
		uv[idx[0]*2+0], uv[idx[1]*2+0], uv[idx[2]*2+0], uv[idx[3]*2+0]
	};
	const uint8_t v[4] = {
		uv[idx[0]*2+1], uv[idx[1]*2+1], uv[idx[2]*2+1], uv[idx[3]*2+1]
	};
	rgb[j*3+0] = clamp(
		(yuv2r(Y[i], u[0], v[0]) + yuv2r(Y[i], u[1], v[1]) +
		 yuv2r(Y[i], u[2], v[2]) + yuv2r(Y[i], u[3], v[3])) / 4.0, 0, 255
	);
	rgb[j*3+1] = clamp(
		(yuv2g(Y[i], u[0], v[0]) + yuv2g(Y[i], u[1], v[1]) +
		 yuv2g(Y[i], u[2], v[2]) + yuv2g(Y[i], u[3], v[3])) / 4.0, 0, 255
	);
	rgb[j*3+2] = clamp(
		(yuv2b(Y[i], u[0], v[0]) + yuv2b(Y[i], u[1], v[1]) +
		 yuv2b(Y[i], u[2], v[2]) + yuv2b(Y[i], u[3], v[3])) / 4.0, 0, 255
	);
}
